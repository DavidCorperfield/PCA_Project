#include "../include/feed_forward.h"
#include <hip/hip_runtime_api.h>

int main(int argc, char **argv){
    printf("testing the feed forward network\n");
    
    //uint8_t *images = get_data("train-images.idx3-ubyte");
    
    //allocate space on the host
    int size_weights = sizeof(float) * NUM_NEURONS*NUM_WEIGHTS;
    float *h_weights = (float*)malloc(size_weights);    
    
    int size_neurons = sizeof(float) * NUM_NEURONS;
    float *h_input = (float*)malloc(size_neurons);
    float *h_output = (float*)malloc(size_neurons);
  
    for(int i = 0; i < NUM_NEURONS; i++){
        h_input[i] = 1;
    }

    for(int i = 0; i < NUM_NEURONS*NUM_WEIGHTS; i++){
        h_weights[i] = 1;
    }
   
    //allocate vectors on the device
    float *d_weights;
    hipMalloc(&d_weights, size_weights);
    float *d_input;
    hipMalloc(&d_input, size_neurons);
    float *d_output;
    hipMalloc(&d_output, size_neurons);
     
    hipError_t error;
    //copy from cpu(host) to the gpu(device)
    error = hipMemcpy(d_weights, h_weights, size_weights, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_input, h_input, size_neurons, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    //evaluate the test layer
    eval_layer<<<NUM_NEURONS, NUM_WEIGHTS>>>(d_input, d_weights, d_output);
    
    //read back the output values from the layer
    hipMemcpy(h_output, d_output, size_neurons, hipMemcpyDeviceToHost);    
    
    printf("test: %f\n" , h_output[0]);
    hipFree(&d_output);hipFree(&d_input);hipFree(d_weights);
    return 0;
    
}
