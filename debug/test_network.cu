#include "../include/backprop.h"
#include "../include/feed_forward.h"
#include "../include/parse_data.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv){
    printf("testing the outer layer using back propagation\n");
    
    uint8_t *images = get_data("train-images.idx3-ubyte");
    uint8_t *labels = get_data("train-labels.idx1-ubyte");
    //uint8_t *labels = images;
    int num_layers = 2;
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
    
    int size_weights = (int)sizeof(float)*MAX_NUM_NEURONS*MAX_NUM_WEIGHTS*num_layers;
    //array to hold EVERY weight value of the network
    float *h_weights = (float*)malloc((size_t)size_weights);    
    
    int size_outputs = (int)sizeof(float)*MAX_NUM_NEURONS*num_layers;
    float *h_outputs = (float*)malloc((size_t)size_outputs);
    
    int size_input = (int)sizeof(float)*MAX_NUM_NEURONS;
    float *h_input = (float*)malloc((size_t)size_input);
    float *h_input2 = (float*)malloc((size_t)size_input);
    float *h_input3 = (float*)malloc((size_t)size_input);
    
    if(h_outputs == NULL){
        printf("unable to create host output pointer");
    }
    for(int i = 0; i < MAX_NUM_NEURONS; i++){
        h_input[i] = (((float)images[i]*1.6)/255.0)-0.8;//scale the input data to -0.8 to 0.8
    }
    for(int i = 0; i < MAX_NUM_NEURONS*MAX_NUM_WEIGHTS*num_layers; i++){
        h_weights[i] = (float)rand()/(float)RAND_MAX;
    }
   
    //allocate vectors on the device
    hipError_t error;
    float *d_weights;
    error = hipMalloc((void**)&d_weights, size_weights);
     if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    float *d_input;
    error = hipMalloc((void **)&d_input, size_input);
     if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    float *d_outputs;
    error = hipMalloc((void **)&d_outputs, size_outputs);
     if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    //copy from cpu(host) to the gpu(device)
    error = hipMemcpy(d_weights, h_weights, size_weights, hipMemcpyHostToDevice); 
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_input, h_input, size_input, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    for(int j = 0; j < size_outputs/sizeof(float);j++){
        h_outputs[j] = (float)0.1;
    }
    error = hipMemcpy(d_outputs, h_outputs, size_outputs, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }    
    
    ///***********Backprop allocations*************/
    int size_network_output = (int)sizeof(float)*(int)NUM_OUTPUT_NEURONS;
    float *h_desired_output = (float*)malloc((size_t)size_network_output);
    
    //set all character outputs to false, or -0.8
    for(int i = 0; i < NUM_OUTPUT_NEURONS; i++){
        h_desired_output[i] = -0.8;
    }
    //set the desired output for the first hand written character to 0.8
    h_desired_output[(int)labels[0]] = 0.8;
    printf("testing with character %i \n", (int)labels[0]);
    
    float *d_desired_output;
    error = hipMalloc((void**)&d_desired_output, (size_t)size_network_output);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_desired_output, h_desired_output, size_network_output, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    
    if(deviceProp.major == 1){
        for(int j = 0; j < 20; j ++){
            //eval_network<<<784, 512>>>(512, num_layers, 784, 784, d_input, d_weights, d_outputs); 
            hipDeviceSynchronize();
            backprop_network<<<784, 512>>>(512, num_layers, 784, 784, d_input, d_outputs, d_desired_output, d_weights); 
            hipDeviceSynchronize();
            //read back the output values from the layer
            hipMemcpy(h_weights, d_weights, size_weights, hipMemcpyDeviceToHost);
            //hipMemcpy(h_desired_output, d_desired_output, size_network_output, hipMemcpyDeviceToHost);
            hipMemcpy(h_outputs, d_outputs, size_outputs, hipMemcpyDeviceToHost);  
            error = hipGetLastError();
            printf("running eval_network returned error code %s, line(%d)\n", hipGetErrorString(error), __LINE__);
           // for(int i = 0; i < 20; i++){
           for(int i = (num_layers-1)*MAX_NUM_NEURONS*MAX_NUM_WEIGHTS; i <(num_layers-1)*MAX_NUM_NEURONS*MAX_NUM_WEIGHTS+20; i++){
                printf("weight%i: %f\n" , i, h_weights[i]);
                
            }
            for(int i = 0; i < 10; i++){
            //for(int i = (num_layers-1)*MAX_NUM_NEURONS; i < (num_layers-1)*MAX_NUM_NEURONS+NUM_OUTPUT_NEURONS; i++){
             //   printf("output%i: %f\n" , i-(num_layers-1)*MAX_NUM_NEURONS, h_outputs[i]);
           //     printf("output%i: %f\n" , i, h_desired_output[i]);
                
            }
        }
    //    eval_layer<<<512, 512>>>(MAX_NUM_WEIGHTS, OUTPUT_LAYER_NEURONS, d_layer_input, d_actual_output, d_desired_output, 
      //                                      d_weights, d_error_prev);
                                            
                                            
    }
    else{
        //eval_layer<<<(int)MAX_NUM_NEURONS, (int)MAX_NUM_WEIGHTS>>>(d_input, d_weights, d_output#include "../include/feed_forward.h"    
    }
        return 0;
        //hipFree(d_actual_output);hipFree(d_layer_input);hipFree(d_desired_output);hipFree(d_weights);
}
