#include "hip/hip_runtime.h"
#include "../include/backprop.h"
#include "../include/feed_forward.h"
#include "../include/parse_data.h"
#include <string.h>
#include "../include/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <time.h>

int main(int argc, char **argv){
    printf("starting the neural network!\n");
    uint8_t *images = get_data("train-images.idx3-ubyte");
    uint8_t *test_images = get_data("t10k-images.idx3-ubyte");
    float *h_images = get_data_f("train-images.idx3-ubyte");
    float *h_images1 = get_data_f("train-images.idx3-ubyte");
    float *h_test_images = get_data_f("t10k-images.idx3-ubyte");
    uint8_t *labels = get_data("train-labels.idx1-ubyte");
    uint8_t *test_labels = get_data("t10k-labels.idx1-ubyte");
    
    //uint8_t *labels = images;    
    int deviceCount;
    hipError_t error;
    hipGetDeviceCount(&deviceCount);
    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
    int num_layers = 3;
    int size_weights = (int)sizeof(float)*MAX_NUM_NEURONS*MAX_NUM_WEIGHTS*num_layers;
    //array to hold EVERY weight value of the network
    float *h_weights = (float*)malloc((size_t)size_weights);    
    
    int size_outputs = (int)sizeof(float)*MAX_NUM_NEURONS*num_layers;
    float *h_outputs = (float*)malloc((size_t)size_outputs);
    
    int size_input = (int)sizeof(float)*MAX_NUM_NEURONS;
    float *h_input = (float*)malloc((size_t)size_input);
    float *h_input2 = (float*)malloc((size_t)size_input);
    float *h_input3 = (float*)malloc((size_t)size_input);
    
    int size_images = size_input*60000;
    int size_test_images = size_input*10000;
    
    clock_t time_normalize = clock();
    printf("about to normalize the images\n");
    /***********************normalize images********************************************************/
    float *d_images;
    //how many images should we normalize at a time? 
    int num_images = 10000;
    int steps = size_images/size_input/num_images;
    int size_step_images = (int)size_input*num_images;
    size_t free1;  
    size_t total;  
    hipMemGetInfo(&free1, &total);  
    printf("memory free is %i\n", (int)free1);
    error = hipMalloc((void**)&d_images, (size_t)(size_step_images));
    size_t free2;
    hipMemGetInfo(&free2, &total);  
   // printf("memory used is %i\n", (int)free1-free2);
    error = hipMalloc((void**)&d_images, (size_t)(size_step_images));
   // printf("size allocated was %f\n", (float)size_step_images/4);
    if (error != hipSuccess){
        printf("hipMalloc returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    //get the amount of free memory on the graphics card  

    for(int i = 0; i < steps; i++){
        error = hipMemcpy(d_images, h_images + i*num_images*MAX_NUM_NEURONS, (size_t)size_step_images, hipMemcpyHostToDevice);
        if (error != hipSuccess){
            printf("hipMalloc returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
            exit(EXIT_FAILURE);
        }
        hipDeviceSynchronize();
        normalize_inputs<<<1,512>>>(512, d_images, MAX_NUM_NEURONS*num_images);
        hipDeviceSynchronize();
        error = hipGetLastError();
        if (error != hipSuccess){
            printf("normalize returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMemcpy(h_images + i*MAX_NUM_NEURONS*num_images, d_images, (size_t)size_step_images, hipMemcpyDeviceToHost);
        if (error != hipSuccess){
            printf("cudamemcopy returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
            exit(EXIT_FAILURE);
        }
    }
    hipFree(d_images);
    
    //normalize the test inputs
    float *d_test_images;
    error = hipMalloc((void**)&d_test_images, (size_t)(size_test_images));
    if (error != hipSuccess){
        printf("hipMalloc returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_test_images, h_test_images, size_test_images, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    normalize_inputs<<<1,512>>>(512, d_test_images, MAX_NUM_NEURONS*num_images);
    
    error = hipMemcpy(h_test_images, d_images, size_test_images, hipMemcpyDeviceToHost);
    if (error != hipSuccess){
        printf("cudamemcopy returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    hipFree(d_test_images);
    /************************************************************************************************/
    
    time_normalize = clock() - time_normalize;
    printf("it took %f seconds to normalize the data\n", time_normalize);
    srand(time(NULL));
    //print_example(0, images, labels);
    for(int i = 0; i < MAX_NUM_NEURONS*MAX_NUM_WEIGHTS*num_layers; i++){
        h_weights[i] = (float)rand()/(float)RAND_MAX*(2*(1/sqrt(MAX_NUM_NEURONS))) - (float)1/sqrt(MAX_NUM_NEURONS);
    }
   
    //allocate vectors on the device
    float *d_weights;
    error = hipMalloc((void**)&d_weights, size_weights);
     if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    float *d_input;
    error = hipMalloc((void **)&d_input, size_input);
     if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    float *d_outputs;
    error = hipMalloc((void **)&d_outputs, size_outputs);
     if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    //copy from cpu(host) to the gpu(device)
    error = hipMemcpy(d_weights, h_weights, size_weights, hipMemcpyHostToDevice); 
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_input, h_input, size_input, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    for(int j = 0; j < size_outputs/sizeof(float);j++){
        h_outputs[j] = (float)0.1;
    }
    error = hipMemcpy(d_outputs, h_outputs, size_outputs, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }    
    
    ///***********Backprop allocations*************/
    int size_network_output = (int)sizeof(float)*(int)NUM_OUTPUT_NEURONS;
    float *h_desired_output = (float*)malloc((size_t)size_network_output);
    
    //set all character outputs to false, or -0.8
    //for(int i = 0; i < NUM_OUTPUT_NEURONS; i++){
    //    h_desired_output[i] = -0.8;
    //}
    //set the desired output for the first hand written character to 0.8
   // h_desired_output[(int)labels[0]] = 0.8;
   // printf("testing with character %i \n", (int)labels[0]);
   // for(int i = 0; i < NUM_OUTPUT_NEURONS; i++){
           // printf("%f", h_desired_output[i]);
   // }
    
    float *d_desired_output;
    error = hipMalloc((void**)&d_desired_output, (size_t)size_network_output);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    //error = hipMemcpy(d_desired_output, h_desired_output, size_network_output, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }
    
    float *d_error_prev;
    float *h_error_prev = (float*)malloc((size_t)size_input);
    error = hipMalloc((void**)&d_error_prev, (size_t)size_input);
    if (error != hipSuccess){
        printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
        exit(EXIT_FAILURE);
    }

/************************************Train the netowork*************************************************************************/
    printf("about to train the network\n");
    if(deviceProp.major == 1){
        int epoch; int j;
        clock_t start = clock();
        clock_t end = clock();
        clock_t kernel_time = (clock_t)0;
        clock_t kernel_start = clock();
        hipDeviceSynchronize();
        for (int test = 0; test < 60000*MAX_NUM_NEURONS; test++){
            //if(h_images[test] != 2){
            if(abs(h_images[test] - ((h_images1[test]*1.6)/255.0 -(float)0.8)) > 0.01){//scale the input data to -0.8 to 0.8
                printf("error at %i\n with h_images = %f and h_images1 = %f and h_images1 unscaled was %f and should be %i\n", test, h_images[test],(h_images1[test]*1.6)/255.0 - (float)0.8, h_images1[test], images[test]);
                test = 60000*MAX_NUM_NEURONS;
            }
        }
        printf("no error\n");
        start = clock();
        for(epoch = 0; epoch < 2; epoch++){
            printf("epoch num: %i\n", epoch);
            for(j = 0; j < 60000; j++) {
               // printf("img is %i\n", labels[j]);
                /******prepare data for next loop*******/
               get_input_image(h_images, h_input, j);
               //get_norm_image(h_input, images, j);
               error = hipMemcpy(d_input, h_input, (size_t)size_input, hipMemcpyHostToDevice);
                if (error != hipSuccess){
                    printf("cudamemcopy d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
                    exit(EXIT_FAILURE);
                }
                //set all character outputs to false, or -0.8
                for(int i = 0; i < NUM_OUTPUT_NEURONS; i++){
                    h_desired_output[i] = -0.8;
                }
                //set the desired output for the first hand written character to 0.8
                h_desired_output[(int)labels[j]] = 0.8;
                //printf("desired output is %i\n", (int)labels[1]);
                error = hipMemcpy(d_desired_output, h_desired_output, size_network_output, hipMemcpyHostToDevice);
                if (error != hipSuccess){
                    printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
                    exit(EXIT_FAILURE);
                }
                
                kernel_start = clock();
                for(int l = 0; l < num_layers-1; l++){
                    eval_layer<<<784, 512>>>(512, l, 784, 784, d_input, d_weights, d_outputs); 
                    hipDeviceSynchronize();
                }
                eval_layer<<<784, 512>>>(512, num_layers-1, 784, 10, d_input, d_weights, d_outputs); 
                hipDeviceSynchronize();
                
                backprop_output_layer<<<784, 512>>>(512, num_layers, 784, 784, d_outputs, d_desired_output, d_weights, d_error_prev); 
                hipDeviceSynchronize();
                
                for(int l = num_layers-2; l >= 0; l--){
                    backprop_layer<<<784, 512>>>(512, l, 784, 784, d_outputs, d_input, d_weights, d_error_prev); 
                    hipDeviceSynchronize();

                }
                kernel_time += clock() - kernel_start;
                //read back the output values from the layer
               // hipMemcpy(h_weights, d_weights, size_weights, hipMemcpyDeviceToHost);
                //hipMemcpy(h_desired_output, d_desired_output, size_network_output, hipMemcpyDeviceToHost);
               // hipMemcpy(h_outputs, d_outputs, size_outputs, hipMemcpyDeviceToHost);  
               // hipMemcpy(h_error_prev, d_error_prev, size_input, hipMemcpyDeviceToHost);  
                error = hipGetLastError();
                if(strcmp(hipGetErrorString(error),"no error"))
                    printf("running eval_network returned error code %s, line(%d)\n", hipGetErrorString(error), __LINE__);
                // for(int i = 0; i < 20; i++){
                /*for(int i = (num_layers-1)*MAX_NUM_NEURONS*MAX_NUM_WEIGHTS; i <(num_layers-1)*MAX_NUM_NEURONS*MAX_NUM_WEIGHTS+20; i++){
                    //printf("weight%i: %f\n" , i, h_weights[i]);
                    
                }
                //for(int i = 0; i < 10; i++){
                for(int i = (num_layers-1)*MAX_NUM_NEURONS; i < (num_layers-1)*MAX_NUM_NEURONS+NUM_OUTPUT_NEURONS; i++){
                //    printf("output%i: %f\n" , i-(num_layers-1)*MAX_NUM_NEURONS, h_outputs[i]);
                //     printf("output%i: %f\n" , i, h_desired_output[i]);
                    }
                //printf("\n");
                for(int i = 0; i < 10; i++){
                    //printf("error%i: %f\n" , i, h_error_prev[i]);
                //     printf("output%i: %f\n" , i, h_desired_output[i]);
                    }*/
                
            }
        }
        end = clock();
        printf("backprop took %f seconds and spent %f seconds on the gpu\n", (float)(end - start)/(float)CLOCKS_PER_SEC, (float)kernel_time/CLOCKS_PER_SEC);
        /**********************test the network***********************************************************************************/
        int final_output; int errors = 0;
        float current_max;
        int test_loops = 10000;
        start = clock();
        kernel_time = 0;
        for(int j=0; j < test_loops; j++){
            get_input_image(h_test_images,h_input, j);
            //get_norm_image(h_input, test_images, j);
            error = hipMemcpy(d_input, h_input, size_input, hipMemcpyHostToDevice);
            if (error != hipSuccess){
                printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
                exit(EXIT_FAILURE);
            }
            //set all character outputs to false, or -0.8
            for(int i = 0; i < NUM_OUTPUT_NEURONS; i++){
                h_desired_output[i] = -0.8;
            }
            //set the desired output for the first hand written character to 0.8
            h_desired_output[(int)test_labels[j]] = 0.8;
            //printf("the desired output is %i\n", test_labels[j]);
            error = hipMemcpy(d_desired_output, h_desired_output, size_network_output, hipMemcpyHostToDevice);
            if (error != hipSuccess){
                printf("hipMalloc d_A returned error %s, line(%d)\n", hipGetErrorString(error), __LINE__);
                exit(EXIT_FAILURE);
            }
            kernel_start = clock();
            for(int l = 0; l < num_layers-1; l++){
                    eval_layer<<<784, 512>>>(512, l, 784, 784, d_input, d_weights, d_outputs); 
                    hipDeviceSynchronize();
                }
            eval_layer<<<784, 512>>>(512, num_layers-1, 784, 10, d_input, d_weights, d_outputs); 
            hipDeviceSynchronize();
            kernel_time += clock() - kernel_start;
            
            hipMemcpy(h_outputs, d_outputs, size_outputs, hipMemcpyDeviceToHost);  
            error = hipGetLastError();
            //printf("running eval_network returned error code %s, line(%d)\n", hipGetErrorString(error), __LINE__);
            current_max = -10;
            for(int k = (int)(num_layers-1)*MAX_NUM_NEURONS; k < (int)(num_layers-1)*MAX_NUM_NEURONS+NUM_OUTPUT_NEURONS; k++){
                //printf("k = %i and output = %f \n", k-MAX_NUM_NEURONS, h_outputs[k]);
                if((float)h_outputs[k] > current_max){
                    current_max = h_outputs[k];
                    final_output = (int)k-(num_layers-1)*MAX_NUM_NEURONS;
                }
            }
            if (final_output != test_labels[j]){
                errors += 1;
                //printf("output should be: %i      out was: %i\n", test_labels[j], final_output);
            }        
        }
        end = clock();
        printf("testing took %f seconds and spent %f seconds on the gpu\n", (float)(end - start)/(float)CLOCKS_PER_SEC, (float)kernel_time/CLOCKS_PER_SEC);
        printf("there were %i errors which makes a percent correct of %f %%\n", errors, 100*(float)(test_loops-errors)/test_loops);                                           
/*************************************************************************************************************************************/
    }
        return 0;
}
