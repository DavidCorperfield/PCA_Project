#include "hip/hip_runtime.h"
#include "../include/backprop.h"

__global__ void  
backprop_output_layer(float *layer_input, float *actual_output, float *desired_output, float *weights, float *error_prev, int num_weights, int num_neurons){
    
    //each block will correspond to one neuron
    int neuron_index = blockIdx.x;
    
    //each thread will correspond to a weight of a neuron
    int weight_index = threadIdx.x;      
    
    __shared__ int error[MAX_NUM_WEIGHTS];
    
    if(weight_index < num_weights and num_neurons == OUTPUT_LAYER_NEURONS){
        //calculate error
        error[weight_index] = actual_output[weight_index] - desired_output[weight_index]);
        //calculate partial derivative error
        error[weight_index] = (1 - layer_output[weight_index]^2) * error[weight_index];
        //calcuate final error for finding the weight change amount, will have to perform a reduction
        error[weight_index] = layer_input[weight_index]*error[weight_index];
        //now calculate errors for the previous layer
        error_prev[weight_index] = weights[weight_index]*error[weight_index];
        
        //re-adjust weights for the current layer
        weights[weight_index] = weights[weight_index] - LEARNING_RATE*error[weight_index];
    }
    
    
    
}
